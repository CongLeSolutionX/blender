/* SPDX-FileCopyrightText: 2021-2022 Blender Foundation
 *
 * SPDX-License-Identifier: Apache-2.0 */

#define CCL_EXTERN_DECLS

#include "kernel/device/cuda/compat.h"
#include "kernel/device/cuda/config.h"
#include "kernel/device/cuda/globals.h"

#include "kernel/tables.h"

#include "kernel/integrator/state.h"
#include "kernel/integrator/state_flow.h"
#include "kernel/integrator/state_util.h"

#include "kernel/integrator/shadow_catcher.h"

#include "kernel/device/gpu/kernel_integrator_shadow_catcher.h"
