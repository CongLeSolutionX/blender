/* SPDX-FileCopyrightText: 2011-2022 Blender Foundation
 *
 * SPDX-License-Identifier: Apache-2.0 */

#include "kernel/device/cuda/compat.h"
#include "kernel/device/cuda/config.h"
#include "kernel/device/cuda/globals.h"

#include "kernel/tables.h"

__constant__ KernelParamsCUDA kernel_params;

#include "kernel/device/gpu/image.h"

#include "kernel/integrator/state.h"
#include "kernel/integrator/state_flow.h"
#include "kernel/integrator/state_util.h"

#include "kernel/bvh/bvh.h"
#include "kernel/geom/motion_triangle_shader.h"
#include "kernel/geom/subd_triangle.h"
#include "kernel/light/distribution.h"
#include "kernel/svm/svm.h"
