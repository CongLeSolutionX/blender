/* SPDX-FileCopyrightText: 2011-2022 Blender Foundation
 *
 * SPDX-License-Identifier: Apache-2.0 */

#include "kernel/device/cuda/compat.h"
#include "kernel/device/cuda/config.h"
#include "kernel/device/cuda/globals.h"

#include "kernel/film/adaptive_sampling.h"

#include "kernel/device/gpu/kernel_adaptive_sampling.h"
