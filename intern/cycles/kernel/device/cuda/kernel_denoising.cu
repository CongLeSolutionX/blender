/* SPDX-FileCopyrightText: 2011-2022 Blender Foundation
 *
 * SPDX-License-Identifier: Apache-2.0 */

#define CCL_EXTERN_DECLS

#include "kernel/device/cuda/compat.h"
#include "kernel/device/cuda/config.h"

#include "kernel/types.h"

#include "kernel/device/gpu/kernel_denoising.h"
