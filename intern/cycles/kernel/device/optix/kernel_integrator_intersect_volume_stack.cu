#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2011-2022 Blender Foundation
 *
 * SPDX-License-Identifier: Apache-2.0 */

#include "kernel/device/optix/compat.h"
#include "kernel/device/optix/globals.h"

#include "kernel/device/gpu/image.h"
#include "kernel/tables.h"

#include "kernel/integrator/state.h"
#include "kernel/integrator/state_flow.h"
#include "kernel/integrator/state_util.h"

#include "kernel/integrator/intersect_volume_stack.h"

extern "C" __global__ void __raygen__kernel_optix_integrator_intersect_volume_stack()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_intersect_volume_stack(nullptr, path_index);
}
