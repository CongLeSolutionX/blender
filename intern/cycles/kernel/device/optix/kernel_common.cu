#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2011-2022 Blender Foundation
 *
 * SPDX-License-Identifier: Apache-2.0 */

#include "kernel/device/optix/compat.h"
#include "kernel/device/optix/globals.h"

#include "kernel/tables.h"

#include "kernel/device/gpu/image.h"

#include "kernel/integrator/state_util.h"

#include "kernel/bvh/bvh.h"
#include "kernel/geom/motion_triangle_shader.h"
#include "kernel/geom/subd_triangle.h"
#include "kernel/svm/svm.h"

// Instantiate SVM node functions
#define INSTANTIATE_FUNCTION1(ret, name, args) \
  template ccl_device_noinline ret name<KERNEL_FEATURE_NODE_MASK_SURFACE, SHADER_TYPE_SURFACE> \
      args; \
  template ccl_device_noinline ret \
      name<KERNEL_FEATURE_NODE_MASK_SURFACE & ~KERNEL_FEATURE_NODE_RAYTRACE, SHADER_TYPE_SURFACE> \
          args; \
  template ccl_device_noinline ret name< \
      (KERNEL_FEATURE_NODE_MASK_SURFACE & ~KERNEL_FEATURE_NODE_RAYTRACE) | KERNEL_FEATURE_MNEE, \
      SHADER_TYPE_SURFACE> \
      args; \
  template ccl_device_noinline ret \
      name<KERNEL_FEATURE_NODE_MASK_SURFACE_LIGHT & \
               ~(KERNEL_FEATURE_NODE_RAYTRACE | KERNEL_FEATURE_NODE_LIGHT_PATH), \
           SHADER_TYPE_SURFACE> \
          args; \
  template ccl_device_noinline ret \
      name<KERNEL_FEATURE_NODE_MASK_SURFACE_SHADOW & \
               ~(KERNEL_FEATURE_NODE_RAYTRACE | KERNEL_FEATURE_NODE_LIGHT_PATH), \
           SHADER_TYPE_SURFACE> \
          args; \
  template ccl_device_noinline ret \
      name<KERNEL_FEATURE_NODE_MASK_SURFACE_BACKGROUND, SHADER_TYPE_SURFACE> \
          args; \
  template ccl_device_noinline ret \
      name<KERNEL_FEATURE_NODE_MASK_SURFACE_LIGHT, SHADER_TYPE_SURFACE> \
          args; \
  template ccl_device_noinline ret \
      name<KERNEL_FEATURE_NODE_MASK_SURFACE_SHADOW, SHADER_TYPE_SURFACE> \
          args; \
  template ccl_device_noinline ret name<KERNEL_FEATURE_NODE_MASK_BUMP, SHADER_TYPE_SURFACE> args; \
  template ccl_device_noinline ret \
      name<KERNEL_FEATURE_NODE_MASK_DISPLACEMENT, SHADER_TYPE_DISPLACEMENT> \
          args; \
  template ccl_device_noinline ret name<KERNEL_FEATURE_NODE_MASK_VOLUME, SHADER_TYPE_VOLUME> args;
#define INSTANTIATE_FUNCTION2(ret, name, args) \
  template ccl_device_noinline ret name<KERNEL_FEATURE_NODE_MASK_SURFACE> args; \
  template ccl_device_noinline ret \
      name<KERNEL_FEATURE_NODE_MASK_SURFACE & ~KERNEL_FEATURE_NODE_RAYTRACE> \
          args; \
  template ccl_device_noinline ret name< \
      (KERNEL_FEATURE_NODE_MASK_SURFACE & ~KERNEL_FEATURE_NODE_RAYTRACE) | KERNEL_FEATURE_MNEE> \
      args; \
  template ccl_device_noinline ret \
      name<KERNEL_FEATURE_NODE_MASK_SURFACE_LIGHT & \
           ~(KERNEL_FEATURE_NODE_RAYTRACE | KERNEL_FEATURE_NODE_LIGHT_PATH)> \
          args; \
  template ccl_device_noinline ret \
      name<KERNEL_FEATURE_NODE_MASK_SURFACE_SHADOW & \
           ~(KERNEL_FEATURE_NODE_RAYTRACE | KERNEL_FEATURE_NODE_LIGHT_PATH)> \
          args; \
  template ccl_device_noinline ret name<KERNEL_FEATURE_NODE_MASK_SURFACE_BACKGROUND> args; \
  template ccl_device_noinline ret name<KERNEL_FEATURE_NODE_MASK_SURFACE_LIGHT> args; \
  template ccl_device_noinline ret name<KERNEL_FEATURE_NODE_MASK_SURFACE_SHADOW> args; \
  template ccl_device_noinline ret name<KERNEL_FEATURE_NODE_MASK_DISPLACEMENT> args; \
  template ccl_device_noinline ret name<KERNEL_FEATURE_NODE_MASK_VOLUME> args;
#define INSTANTIATE_FUNCTION3(ret, name, args) \
  template ccl_device_noinline ret name<SHADER_TYPE_SURFACE> args; \
  template ccl_device_noinline ret name<SHADER_TYPE_DISPLACEMENT> args; \
  template ccl_device_noinline ret name<SHADER_TYPE_VOLUME> args;

INSTANTIATE_FUNCTION1(int,
                      svm_node_closure_bsdf,
                      (KernelGlobals kg,
                       ccl_private ShaderData *sd,
                       ccl_private float *stack,
                       Spectrum closure_weight,
                       uint4 node,
                       uint32_t path_flag,
                       int offset))
INSTANTIATE_FUNCTION2(
    void,
    svm_node_attr,
    (KernelGlobals kg, ccl_private ShaderData *sd, ccl_private float *stack, uint4 node))
INSTANTIATE_FUNCTION2(
    void,
    svm_node_set_displacement,
    (KernelGlobals kg, ccl_private ShaderData *sd, ccl_private float *stack, uint fac_offset))
INSTANTIATE_FUNCTION2(
    void,
    svm_node_displacement,
    (KernelGlobals kg, ccl_private ShaderData *sd, ccl_private float *stack, uint4 node))
INSTANTIATE_FUNCTION2(int,
                      svm_node_vector_displacement,
                      (KernelGlobals kg,
                       ccl_private ShaderData *sd,
                       ccl_private float *stack,
                       uint4 node,
                       int offset))
INSTANTIATE_FUNCTION2(
    void,
    svm_node_set_bump,
    (KernelGlobals kg, ccl_private ShaderData *sd, ccl_private float *stack, uint4 node))
INSTANTIATE_FUNCTION3(void,
                      svm_node_closure_volume,
                      (KernelGlobals kg,
                       ccl_private ShaderData *sd,
                       ccl_private float *stack,
                       Spectrum closure_weight,
                       uint4 node))
INSTANTIATE_FUNCTION3(int,
                      svm_node_principled_volume,
                      (KernelGlobals kg,
                       ccl_private ShaderData *sd,
                       ccl_private float *stack,
                       Spectrum closure_weight,
                       uint4 node,
                       uint32_t path_flag,
                       int offset))
INSTANTIATE_FUNCTION2(void,
                      svm_node_light_path,
                      (KernelGlobals kg,
                       ConstIntegratorState state,
                       ccl_private const ShaderData *sd,
                       ccl_private float *stack,
                       uint type,
                       uint out_offset,
                       uint32_t path_flag))
INSTANTIATE_FUNCTION2(int,
                      svm_node_tex_voronoi,
                      (KernelGlobals kg,
                       ccl_private ShaderData *sd,
                       ccl_private float *stack,
                       uint dimensions,
                       uint feature,
                       uint metric,
                       int offset))
#if defined(__SHADER_RAYTRACE__)
INSTANTIATE_FUNCTION2(void,
                      svm_node_bevel,
                      (KernelGlobals kg,
                       ConstIntegratorState state,
                       ccl_private ShaderData *sd,
                       ccl_private float *stack,
                       uint4 node))
INSTANTIATE_FUNCTION2(void,
                      svm_node_ao,
                      (KernelGlobals kg,
                       ConstIntegratorState state,
                       ccl_private ShaderData *sd,
                       ccl_private float *stack,
                       uint4 node))
#endif
INSTANTIATE_FUNCTION2(int,
                      svm_node_tex_voxel,
                      (KernelGlobals kg,
                       ccl_private ShaderData *sd,
                       ccl_private float *stack,
                       uint4 node,
                       int offset))
