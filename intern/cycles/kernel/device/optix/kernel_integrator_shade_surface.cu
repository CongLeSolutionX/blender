#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2011-2022 Blender Foundation
 *
 * SPDX-License-Identifier: Apache-2.0 */

#define CCL_EXTERN_DECLS

#include "kernel/device/optix/compat.h"
#include "kernel/device/optix/globals.h"

#include "kernel/device/gpu/image.h"
#include "kernel/tables.h"

#include "kernel/integrator/state.h"
#include "kernel/integrator/state_flow.h"
#include "kernel/integrator/state_util.h"

#include "kernel/integrator/shade_surface.h"

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_surface()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_surface(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_surface_raytrace()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_surface_raytrace(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_surface_mnee()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_shade_surface_mnee(nullptr, path_index, kernel_params.render_buffer);
}
