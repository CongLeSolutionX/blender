#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2011-2022 Blender Foundation
 *
 * SPDX-License-Identifier: Apache-2.0 */

#define __OSL__

#include "kernel/device/optix/compat.h"
#include "kernel/device/optix/globals.h"

#include "kernel/device/gpu/image.h"
#include "kernel/tables.h"

#include "kernel/integrator/state.h"
#include "kernel/integrator/state_flow.h"
#include "kernel/integrator/state_util.h"

#include "kernel/bake/bake.h"

extern "C" __global__ void __raygen__kernel_optix_shader_eval_displace()
{
  KernelShaderEvalInput *const input = (KernelShaderEvalInput *)kernel_params.path_index_array;
  float *const output = kernel_params.render_buffer;
  const int global_index = kernel_params.offset + optixGetLaunchIndex().x;
  kernel_displace_evaluate(nullptr, input, output, global_index);
}

extern "C" __global__ void __raygen__kernel_optix_shader_eval_background()
{
  KernelShaderEvalInput *const input = (KernelShaderEvalInput *)kernel_params.path_index_array;
  float *const output = kernel_params.render_buffer;
  const int global_index = kernel_params.offset + optixGetLaunchIndex().x;
  kernel_background_evaluate(nullptr, input, output, global_index);
}

extern "C" __global__ void __raygen__kernel_optix_shader_eval_curve_shadow_transparency()
{
  KernelShaderEvalInput *const input = (KernelShaderEvalInput *)kernel_params.path_index_array;
  float *const output = kernel_params.render_buffer;
  const int global_index = kernel_params.offset + optixGetLaunchIndex().x;
  kernel_curve_shadow_transparency_evaluate(nullptr, input, output, global_index);
}
