#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2011-2022 Blender Foundation
 *
 * SPDX-License-Identifier: Apache-2.0 */

#define CCL_EXTERN_DECLS

#include "kernel/device/optix/compat.h"
#include "kernel/device/optix/globals.h"

#include "kernel/device/gpu/image.h"
#include "kernel/tables_extern.h"

#include "kernel/integrator/state_util.h"

#include "kernel/integrator/intersect_dedicated_light.h"

extern "C" __global__ void __raygen__kernel_optix_integrator_intersect_dedicated_light()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_intersect_dedicated_light(nullptr, path_index);
}
